#include "hip/hip_runtime.h"
// -*- c++ -*-
//
// This file is part of the course materials for AMATH483/583 at the University of Washington,
// Spring 2020
//
// Licensed under Creative Commons Attribution-NonCommercial-ShareAlike 4.0 International License
// https://creativecommons.org/licenses/by-nc-sa/4.0/
//
// Author: Andrew Lumsdaine
//

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <cmath>
#include <cstddef>
#include <iomanip>
#include <iostream>
#include <random>
#include <string>
#include <vector>

#include "Timer.hpp"


template<typename T>
void randomize(std::vector<T>& v) {
  static std::default_random_engine             generator;
  static std::uniform_real_distribution<T> distribution(-1.0, 1.0);
  static auto                                   dice = std::bind(distribution, generator);

  for (size_t i = 0; i < v.size(); ++i) {
    v[i] = dice();
  }
}


template<typename T>
T two_norm_sequential(const std::vector<T>& v) {
  T sum = 0.0;

  for (size_t i = 0; i < v.size(); ++i) {
    sum += v[i] * v[i];
  }

  return std::sqrt(sum);
}



template<typename T>
T norm_thrust(const thrust::device_vector<T>& x) {
  T sum = thrust::reduce(x.begin(), x.end(), 0);
  return std::sqrt(sum);
}


void header(const std::string& msg = "") {
  auto& os_ = std::cout;
  if (msg != "") {
    os_ << std::setw(12) << std::left << msg << std::endl;
  }
  os_ << std::setw(12) << std::right << "N";
  os_ << std::setw(12) << std::right << "Sequential";

  os_ << std::setw(12) << std::right << "First";
  os_ << std::setw(12) << std::right << "Second";

  os_ << std::setw(12) << std::right << "First";
  os_ << std::setw(12) << std::right << "Second";

  os_ << std::endl;
}

double Gflops_sec(size_t nnz, size_t trials, double msec) {
  double Gflops = static_cast<double>(trials) * (2.0 * nnz) / 1.e9;
  double sec    = msec / 1.e3;
  if (sec == 0) {
    return 0;
  }
  return Gflops / sec;
}

void log(size_t nnz, size_t ntrials, const std::vector<double>& ms_times, const std::vector<double>& norms) {
  auto& os_ = std::cout;
  os_ << std::setw(12) << std::right << nnz;

  for (size_t i = 0; i < ms_times.size(); ++i) {
    os_ << std::setw(12) << std::right << Gflops_sec(nnz, ntrials, ms_times[i]);
  }
  for (size_t i = 1; i < ms_times.size(); ++i) {
    os_ << std::setw(14) << std::right << std::abs(norms[i] - norms[0]) / norms[0];
  }
  os_ << std::endl;
}

size_t num_trials(size_t nnz) {
  double N_1k = std::ceil(2E9 / static_cast<double>(nnz));
  return 5 + static_cast<size_t>(N_1k);
}

template <class T, typename Function>
void run_cu(Function&& f, size_t N_min, size_t N_max) {
  header(sizeof(T) == 4 ? "\nFloat" : "\nDouble");
  Timer t;

  for (size_t size = N_min; size <= N_max; size *= 2) {
    std::vector<double> ms_times;
    std::vector<double> norms;

    std::vector<T> x(size);
      
    randomize(x);

    double norm0 = two_norm_sequential(x);
    double norm1 = 0.0;

    size_t ntrials = num_trials(size);

    t.start();
    for (size_t i = 0; i < ntrials; ++i) {
      norm0 = two_norm_sequential(x);
    }
    t.stop();
    ms_times.push_back(t.elapsed());
    norms.push_back(norm0);

    thrust::device_vector<T> X(size);
    thrust::copy(x.begin(), x.end(), X.begin());

    for (size_t trip = 0; trip < 2; ++trip) {

      t.start();
      hipDeviceSynchronize();
      for (size_t i = 0; i < ntrials; ++i) {
        norm1 = f(X);
        hipDeviceSynchronize();
      }
      t.stop();
      ms_times.push_back(t.elapsed());
      norms.push_back(norm1);
    }

    log(size, ntrials, ms_times, norms);
  }
}




int main(int argc, char* argv[]) {
  size_t N_min = 1024 * 1024;
  size_t N_max = 128 * 1024 * 1024;

  if (argc >= 2) {
    N_min = std::stol(argv[1]);
  }
  if (argc >= 3) {
    N_max = std::stol(argv[2]);
  }

  run_cu<float>(norm_thrust<float>, N_min, N_max);
  run_cu<double>(norm_thrust<double>, N_min, N_max);

  return 0;
}
